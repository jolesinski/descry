#include "hip/hip_runtime.h"
#include <descry/cupcl/utils.cuh>
#include <descry/cupcl/eigen.cuh>
#include <descry/cupcl/support.cuh>
#include <descry/cupcl/unique.h>

#include <>

#include <thrust/device_vector.h>
#include <pcl/point_types.h>

__host__ __device__ void
fitPlane(const int query_idx, const pcl::PointXYZ* in,
         const int width, const int height,
         const float* projection,
         const float radius, pcl::Normal& normal) {
  Eigen::Matrix3f scatter = Eigen::Matrix3f::Identity();
  if (!isfinite(in[query_idx].x) ||
      !getSupportCovariance(query_idx, in, width, height, projection, radius, scatter)) {
    normal.normal_x = NAN;
    normal.normal_y = NAN;
    normal.normal_z = NAN;
    return;
  }

  Eigen::Vector3f normal_vec, lambdas;
  eigenvalsSymm3x3(scatter, lambdas);
  eigenvecSymm3x3(scatter, lambdas, 2, normal_vec);

  if (lambdas.nonZeros())
    normal.curvature = lambdas(2) / lambdas.sum();
  else
    normal.curvature = NAN;

  // orient towards viewpoint
  if( normal_vec.dot(getVector3f(in[query_idx])) > 0 )
    normal_vec = -normal_vec;

  normal.normal_x = normal_vec(0);
  normal.normal_y = normal_vec(1);
  normal.normal_z = normal_vec(2);
}


__global__ void
computeNormalsKernel(const pcl::PointXYZ* in,
                     const int width, const int height,
                     const float* projection,
                     const float radius,
                     pcl::Normal* out) {
  int u = blockIdx.x * blockDim.x + threadIdx.x;
  int v = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = width*v + u;

  fitPlane(idx, in, width, height, projection, radius, out[idx]);
}

std::unique_ptr<thrust::device_vector<pcl::Normal>> computeNormals() {
  auto normals = std::make_unique<thrust::device_vector<pcl::Normal>>();
  return normals;
};